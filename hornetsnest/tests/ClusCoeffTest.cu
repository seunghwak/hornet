/**
 * @brief
 * @file
 */

#include "HornetAlg.hpp"
#include <StandardAPI.hpp>
#include <Core/GPUCsr/Csr.cuh>
#include <Core/GPUHornet/Hornet.cuh>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>

#include "Static/ClusteringCoefficient/cc.cuh"

using namespace timer;
using namespace hornets_nest;

using HornetGraph = gpu::Hornet<EMPTY, EMPTY>;

int exec(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
    //hornet_graph.print();
    ClusteringCoefficient cc(hornet_graph);
    cc.init();

    Timer<DEVICE> TM(5);
    TM.start();

    cc.run();

    TM.stop();
    TM.print("Computation time:");
  
    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

