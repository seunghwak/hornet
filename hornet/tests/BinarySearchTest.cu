#include "hip/hip_runtime.h"
#include "Host/Numeric.hpp"
#include "Device/Util/DeviceProperties.cuh"
#include "Device/Util/PrintExt.cuh"
#include "Device/Util/Algorithm.cuh"
#include "Device/Primitives/BinarySearchLB.cuh"
#include "Device/Primitives/impl/BinarySearchLB2.i.cuh"
#include "Device/Primitives/MergePathLB.cuh"
#include "Device/Util/Timer.cuh"
//#include <Graph/GraphBase.hpp>
#include <Graph/GraphStd.hpp>
#include <Graph/GraphWeight.hpp>
#include <Graph/BellmanFord.hpp>
#include <Graph/Dijkstra.hpp>

#include <iostream>

#include "Device/Util/Timer.cuh"
#include "Device/DataMovement/impl/Block.i.cuh"
#include <hip/hip_cooperative_groups.h>
//#define ENABLE_MGPU
#include <random>
#include <chrono>
#include "StandardAPI.hpp"

#if defined(ENABLE_MGPU)
    #include <moderngpu/kernel_load_balance.hxx>
#endif

using namespace graph;
using namespace timer;
using namespace hornets_nest;

template<int ITEMS_PER_BLOCK, int BLOCK_SIZE>
__global__
void MergePathTest2(const int* __restrict__ d_partitions,
                    int                     num_partitions,
                    const int* __restrict__ d_prefixsum,
                    int                     prefixsum_size,
                    int* __restrict__       d_pos,
                    int* __restrict__       d_offset) {
    __shared__ int smem[ITEMS_PER_BLOCK];

    const auto& lambda = [&](int pos, int, int index) {
                             d_pos[index] = pos;
                             //d_offset[index] = offset;
                        };
    //xlib::binarySearchLB2<BLOCK_SIZE, ITEMS_PER_BLOCK / BLOCK_SIZE, true>
    //    (d_partitions, num_partitions, d_prefixsum, prefixsum_size, smem, lambda);

    xlib::mergePathLB<BLOCK_SIZE, ITEMS_PER_BLOCK>
        (d_partitions, num_partitions, d_prefixsum, prefixsum_size, smem, lambda);
}

#if 0//used by (commented out) unrechable code, may better be deleted unless this code will be reused.
const bool PRINT      = false;
const int  BLOCK_SIZE = 128;
#endif

__device__ int d_value;

template<int ITEMS_PER_BLOCK, int BLOCK_SIZE>
__global__
void copyKernel(const int* __restrict__ input, int num_blocks, int smem_size) {
    __shared__ int smem[ITEMS_PER_BLOCK];

    for (int i = blockIdx.x; i < num_blocks; i += gridDim.x) {
        xlib::block::StrideOp<0, ITEMS_PER_BLOCK, BLOCK_SIZE>
            ::copy(input + i * ITEMS_PER_BLOCK, smem_size, smem);
        /*auto smem_tmp = smem + threadIdx.x;
        auto d_tmp    = input + i * ITEMS_PER_BLOCK + threadIdx.x;

        #pragma unroll
        for (int i = 0; i < ITEMS_PER_BLOCK; i += BLOCK_SIZE)
            smem_tmp[i] = (i + threadIdx.x < smem_size) ? d_tmp[i] : 0;*/

        if (threadIdx.x > 1023)
            d_value = smem[threadIdx.x];
    }
}


template<int ITEMS_PER_BLOCK, int BLOCK_SIZE>
__global__
void copyKernel2(const int* __restrict__ input, int num_blocks, int smem_size) {
    //__shared__ int smem[ITEMS_PER_BLOCK];

    for (int i = blockIdx.x; i < num_blocks; i += gridDim.x) {
        auto smem_tmp = xlib::dyn_smem + threadIdx.x;
        auto d_tmp    = input + i * ITEMS_PER_BLOCK + threadIdx.x;

        for (int i = threadIdx.x; i < smem_size; i += BLOCK_SIZE) {
            *smem_tmp = *d_tmp;
            smem_tmp += BLOCK_SIZE;
            d_tmp    += BLOCK_SIZE;
        }

        if (threadIdx.x > 1023)
            d_value = xlib::dyn_smem[threadIdx.x];
    }
}


__global__
void noLambdaKernel(const int* __restrict__ ptr2, int* __restrict__ ptr1, int size) {
    int id     = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < size; i += stride) {
        ptr1[i] = ptr2[i];
        ptr1[i + 10] = ptr2[i + 10];
        ptr1[i + 20] = ptr2[i + 20];
    }
}

template<typename Lambda>
__global__
void lambdaKernel(Lambda lambda, int size) {
    int id     = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < size; i += stride)
        lambda(i);
}


template<typename Lambda, typename... TArgs>
__global__
void lambdaKernel2(Lambda lambda, int size, TArgs* __restrict__ ... args) {
    int id     = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < size; i += stride)
        lambda(i, args...);
}

struct LL {
    int*       __restrict__ ptr1;
    const int* __restrict__ ptr2;

    __device__ __forceinline__
    void operator()(int i) {
        const int* __restrict__ vv2 = ptr2;
        int*       __restrict__ vv1 = ptr1;

        vv1[i] = vv2[i];
        vv1[i + 10] = vv2[i + 10];
        vv1[i + 20] = vv2[i + 20];
    }
};

int exec(int argc, char* argv[]) {
    using namespace graph;

    GraphStd<int, int> graph1;
    graph1.read(argv[1]);

    graph1.print_degree_distrib();
    graph1.print_analysis();

    auto weights = new int[graph1.nV()];
    auto seed = std::chrono::high_resolution_clock::now().time_since_epoch()
                .count();
    std::mt19937 engine(seed);
    std::uniform_int_distribution<int> distrib(0, 100);
    std::generate(weights, weights + graph1.nV(),
                  [&](){ return distrib(engine); } );

    GraphWeight<int, int, int> graph_weight(graph1.csr_out_edges(), graph1.nV(),
                                            graph1.csr_out_edges(), graph1.nE(),
                                            weights);


    Timer<HOST> TM1;

    Dijkstra<int, int, int> dijkstra(graph_weight);

    TM1.start();

    for (int i = 0; i < graph1.nV(); i++) {
        dijkstra.run(i);
        dijkstra.reset();
    }
    TM1.stop();
    TM1.print("Dijkstra");

    return 1;
#if 0//unrechable code, may better be deleted unless this code will be reused.
    const int THREAD_ITEMS    = 11;
    const int ITEMS_PER_BLOCK = BLOCK_SIZE * THREAD_ITEMS;

    int num_blocks_copy = 100000;

    int* d_input;
    gpu::allocate(d_input, ITEMS_PER_BLOCK * num_blocks_copy);

    Timer<DEVICE, micro> TM;
    TM.start();

    copyKernel<ITEMS_PER_BLOCK, BLOCK_SIZE>
        <<< num_blocks_copy, BLOCK_SIZE >>> (d_input, num_blocks_copy, 9 * BLOCK_SIZE);

    TM.stop();
    TM.print("copy1");
    TM.start();

    copyKernel2<ITEMS_PER_BLOCK, BLOCK_SIZE>
        <<< num_blocks_copy, BLOCK_SIZE >>> (d_input, num_blocks_copy, 9 * BLOCK_SIZE);

    TM.stop();
    TM.print("copy2");

    gpu::free(d_input);

    return 1;

    GraphStd<> graph;
    graph.read(argv[1], parsing_prop::PRINT_INFO | parsing_prop::RM_SINGLETON);

    int  size       = graph.nV();
    auto prefixsum  = graph.csr_out_offsets();
    int  ceil_total = xlib::upper_approx(graph.nE(), ITEMS_PER_BLOCK);
    //--------------------------------------------------------------------------
    //   HOST
    auto h_pos    = new int[ceil_total];
    auto h_offset = new int[ceil_total];
    for (int i = 0, k = 0; i < size; i++) {
        for (int j = prefixsum[i]; j < prefixsum[i + 1]; j++) {
            h_pos[k]      = i;
            h_offset[k++] = j - prefixsum[i];
        }
    }
    for (int i = prefixsum[size]; i < ceil_total; i++)
        h_pos[i] = -1;
    //--------------------------------------------------------------------------
    int num_merge = graph.nE() + graph.nV();

    if (PRINT) {
        graph.print_raw();
        std::cout << "Experted results:\n\n";
        host::printArray(prefixsum, size + 1);
        host::printArray(h_pos, prefixsum[size]);
        host::printArray(h_offset, prefixsum[size]);
    }

    int* d_prefixsum, *d_pos, *d_offset, *d_partitions;
    int merge_blocks           = xlib::ceil_div<ITEMS_PER_BLOCK>(num_merge);
    int merge_block_partitions = xlib::ceil_div<BLOCK_SIZE>(merge_blocks);

    int num_blocks           = xlib::ceil_div<ITEMS_PER_BLOCK>(graph.nE());
    int num_block_partitions = xlib::ceil_div<BLOCK_SIZE>(num_blocks);

    std::cout <<   "   THREAD_ITEMS:    " << THREAD_ITEMS
              << "\n   ITEMS_PER_BLOCK: " << ITEMS_PER_BLOCK
              << "\n   Total items:     " << graph.nE()
              << "\n   Num blocks:      " << num_blocks
              << "\n   Num Merges Part.: " << merge_blocks
              << "\n" << std::endl;

    gpu::allocate(d_prefixsum, size + 1);
    gpu::allocate(d_pos, ceil_total);
    gpu::allocate(d_offset, ceil_total);
    gpu::allocate(d_partitions, merge_blocks + 1);
    host::copyToDevice(prefixsum, size + 1, d_prefixsum);
    gpu::memsetZero(d_pos, ceil_total);
    gpu::memsetZero(d_offset, ceil_total);
    gpu::memsetZero(d_partitions, num_blocks + 1);
    //--------------------------------------------------------------------------
    TM.start();

    xlib::mergePathLBPartition <ITEMS_PER_BLOCK>
        <<< merge_block_partitions, BLOCK_SIZE >>>
        (d_prefixsum, size, graph.nE(), num_merge, d_partitions, merge_blocks);

    TM.stop();
    TM.print("Partition:  ");

    TM.start();

    MergePathTest2<ITEMS_PER_BLOCK, BLOCK_SIZE> <<< merge_blocks, BLOCK_SIZE >>>
        (d_partitions, merge_blocks, d_prefixsum, size + 1, d_pos, d_offset);

    TM.stop();
    TM.print("BinarySearch:  ");

    CHECK_CUDA_ERROR
    //--------------------------------------------------------------------------
    if (PRINT) {
        std::cout << "Results:\n\n";
        gpu::printArray(d_pos,    graph.nE());
        gpu::printArray(d_offset, graph.nE());
    }

    std::cout << "\n Check Positions: "
              << gpu::equal(h_pos, h_pos + graph.nE(), d_pos)
              //<< "\n   Check Offsets: "
              //<< gpu::equal(h_offset, h_offset + graph.nE(), d_offset)
              << "\n" << std::endl;

    //L1:

#if defined(ENABLE_MGPU)
    using namespace mgpu;
    standard_context_t context;

    int    num_segments = graph.nV();
    int           count = graph.nE();
    const auto&  vector = std::vector<int>(prefixsum, prefixsum + num_segments);
    mem_t<int> segments = to_mem(vector, context);

    mem_t<int> lbs(count, context);
    TM.start();

    load_balance_search(count, segments.data(), num_segments, lbs.data(),
                        context);

    TM.stop();
    TM.print("ModernGPU:  ");

    auto lbs_host = from_mem(lbs);
    std::cout << "\n   Check Offsets: "
              << std::equal(h_pos, h_pos + graph.nE(), lbs_host.data())
              << "\n" << std::endl;
#endif

    gpu::free(d_partitions, d_offset, d_pos, d_prefixsum);

    return 0;
#endif
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

