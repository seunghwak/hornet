#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <gtest/gtest.h>

#include "StandardAPI.hpp"

#include "hornet_test_fixtures.h"

namespace {

__global__ void WarpScanTest() {
    int value = 1;

    __shared__ hipcub::WarpScan<decltype(value)>::TempStorage temp_storage;

    hipcub::WarpScan<decltype(value)>(temp_storage).ExclusiveSum(value, value);
}

__global__ void WarpReduceTest() {
    int value = 1;

    __shared__ hipcub::WarpReduce<decltype(value)>::TempStorage temp_storage;

    auto aggregate = hipcub::WarpReduce<decltype(value)>(temp_storage).Sum(value);
    (void)aggregate;
}

}

class WarpTest : public HornetTest {
protected:
};

TEST_F(WarpTest, WarpScanTest) {
    int num_devices = 0;

    ASSERT_EQ(hipGetDeviceCount(&num_devices), hipSuccess);
    std::cout << "# GPUs=" << num_devices << std::endl;

    for (size_t i = 0; i < static_cast<size_t>(num_devices); ++i) {
        int warp_size = 0;
        ASSERT_EQ(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, i), hipSuccess);
        ASSERT_EQ(hipSetDevice(i), hipSuccess);

        std::cout << "run test on GPU " << i << " warp size=" << warp_size << std::endl;
        WarpScanTest<<<1, warp_size>>>();
    }
}

TEST_F(WarpTest, WarpReduceTest) {
    int num_devices = 0;

    ASSERT_EQ(hipGetDeviceCount(&num_devices), hipSuccess);
    std::cout << "# GPUs=" << num_devices << std::endl;

    for (size_t i = 0; i < static_cast<size_t>(num_devices); ++i) {
        int warp_size = 0;
        ASSERT_EQ(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, i), hipSuccess);
        ASSERT_EQ(hipSetDevice(i), hipSuccess);

        std::cout << "run test on GPU " << i << " warp size=" << warp_size << std::endl;
        WarpReduceTest<<<1, warp_size>>>();
    }
}

