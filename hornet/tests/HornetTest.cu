#include "hip/hip_runtime.h"
#include "Hornet.hpp"
#include "StandardAPI.hpp"
#include "Core/GPUHornet/BatchUpdate.cuh"
#include "Util/BatchFunctions.hpp"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>          //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <hip/hip_runtime_api.h>
//nvprof --profile-from-start off --log-file log.txt --print-gpu-trace

using namespace hornets_nest;
using namespace timer;
using namespace std::string_literals;

using HornetGPU = hornets_nest::gpu::Hornet<EMPTY, EMPTY>;

/**
 * @brief Example tester for Hornet
 */
int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    xlib::device_info();

    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1]);
    auto weights = new int[graph.nE()];
    std::iota(weights, weights + graph.nE(), 0);
    //--------------------------------------------------------------------------
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGPU hornet_gpu(hornet_init);
    std::cout << "------------------------------------------------" <<std::endl;
    //--------------------------------------------------------------------------
    using namespace batch_gen_property;

    if (argc == 3) {
        int batch_size = std::stoi(argv[2]);

#ifdef TEST
        batch_size = 100;
#endif
        vid_t* batch_src, *batch_dst;
        host::allocatePageLocked(batch_src, batch_size);
        host::allocatePageLocked(batch_dst, batch_size);
#ifdef TEST
        for (int i = 0; i < batch_size - 10; ++i) {
            batch_src[i] = 33;
            batch_dst[i] = 8;
        }
        for (int i = batch_size - 10; i < batch_size; ++i) {
            batch_src[i] = 33;
            batch_dst[i] = 8;
        }
#else
        generateBatch(graph, batch_size, batch_src, batch_dst,
                      BatchGenType::INSERT, UNIQUE);
#endif
        gpu::BatchUpdate batch_update(batch_src, batch_dst, batch_size);

        //batch_update.print();
        std::cout << "------------------------------------------------" <<std::endl;

        using namespace gpu::batch_property;

        hornet_gpu.reserveBatchOpResource(batch_size);

        hornet_gpu.print();
        std::cout << "------------------------------------------------" <<std::endl;
        hipProfilerStart();
        Timer<DEVICE> TM(3);
        TM.start();

        hornet_gpu.insertEdgeBatch(batch_update);
        //hornet_gpu.deleteEdgeBatch(batch_update);

        TM.stop();
        //TM.print("Insertion "s + std::to_string(batch_size) + ":  ");
        hipProfilerStop();
        //hornet_gpu.check_sorted_adjs();
        //delete[] batch_src;
        //delete[] batch_dst;
        host::freePageLocked(batch_src, batch_dst);
        //batch_update.print();
        hornet_gpu.print();
    }
    delete[] weights;

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

